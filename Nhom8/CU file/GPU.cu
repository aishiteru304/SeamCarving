#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#define CHECK(call)\
{\
    const hipError_t error = call;\
    if (error != hipSuccess)\
    {\
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);\
        fprintf(stderr, "code: %d, reason: %s\n", error,\
                hipGetErrorString(error));\
        exit(EXIT_FAILURE);\
    }\
}
void printDeviceInfo()
{
	hipDeviceProp_t devProv;
    CHECK(hipGetDeviceProperties(&devProv, 0));
    printf("**********GPU info**********\n");
    printf("Name: %s\n", devProv.name);
    printf("Compute capability: %d.%d\n", devProv.major, devProv.minor);
    printf("Num SMs: %d\n", devProv.multiProcessorCount);
    printf("Max num threads per SM: %d\n", devProv.maxThreadsPerMultiProcessor); 
    printf("Max num warps per SM: %d\n", devProv.maxThreadsPerMultiProcessor / devProv.warpSize);
    printf("GMEM: %lu bytes\n", devProv.totalGlobalMem);
    printf("CMEM: %lu bytes\n", devProv.totalConstMem);
    printf("L2 cache: %i bytes\n", devProv.l2CacheSize);
    printf("SMEM / one SM: %lu bytes\n", devProv.sharedMemPerMultiprocessor);
    printf("****************************\n");
}
struct GpuTimer
{
    hipEvent_t start;
    hipEvent_t stop;

    GpuTimer()
    {
        hipEventCreate(&start);
        hipEventCreate(&stop);
    }

    ~GpuTimer()
    {
        hipEventDestroy(start);
        hipEventDestroy(stop);
    }

    void Start()
    {
        hipEventRecord(start, 0);
        hipEventSynchronize(start);
    }

    void Stop()
    {
        hipEventRecord(stop, 0);
    }

    float Elapsed()
    {
        float elapsed;
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsed, start, stop);
        return elapsed;
    }
};
void readPnm(char * fileName, int &width, int &height, uchar3 * &pixels)
{
	FILE * f = fopen(fileName, "r");
	if (f == NULL)
	{
		printf("Cannot read %s\n", fileName);
		exit(EXIT_FAILURE);
	}

	char type[3];
	fscanf(f, "%s", type);
	
	if (strcmp(type, "P3") != 0) // In this exercise, we don't touch other types
	{
		fclose(f);
		printf("Cannot read %s\n", fileName); 
		exit(EXIT_FAILURE); 
	}

	fscanf(f, "%i", &width);
	fscanf(f, "%i", &height);
	
	int max_val;
	fscanf(f, "%i", &max_val);
	if (max_val > 255) // In this exercise, we assume 1 byte per value
	{
		fclose(f);
		printf("Cannot read %s\n", fileName); 
		exit(EXIT_FAILURE); 
	}

	pixels = (uchar3 *)malloc(width * height * sizeof(uchar3));
	for (int i = 0; i < width * height; i++)
		fscanf(f, "%hhu%hhu%hhu", &pixels[i].x, &pixels[i].y, &pixels[i].z);

	fclose(f);
}
void writePnm(uchar3 * pixels, int width, int height, char * fileName)
{
	FILE * f = fopen(fileName, "w");
	if (f == NULL)
	{
		printf("Cannot write %s\n", fileName);
		exit(EXIT_FAILURE);
	}	

	fprintf(f, "P3\n%i\n%i\n255\n", width, height); 

	for (int i = 0; i < width * height; i++)
		fprintf(f, "%hhu\n%hhu\n%hhu\n", pixels[i].x, pixels[i].y, pixels[i].z);
	
	fclose(f);
}
__global__ void Convolution_Kernel(uchar3 * inPixels, int width, int height, float* filter,
        int filterWidth, 
        uchar3 * outPixels)
{
	//This shared memory contain all memory need to use for one block from global memory
	// Get 2d coordinate inside each thread
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	
	//Filter all valid coordinate
	if (x <= (height - 1) && y <= (width - 1))
	{
		float r = 0;
		float g = 0;
		float b = 0;
		//Loop through all elemnet in kernel
		for (int i = 0; i < filterWidth * filterWidth; i++)
		{
			// Convert it back to 2d coordinate
			// This is pixel aroung center (x, y) that we do the convolution
			int relative_x = x + int(i/filterWidth) - int(filterWidth/2);
			int relative_y = y + i%filterWidth - int(filterWidth/2);
			// Handle Boundary convolution ( Padding technique )
			if (relative_x <= -1) 
			{
				relative_x = 0;
			}
			if (relative_x >= width)
			{
				relative_x = width - 1;
			}
			if (relative_y <= -1) 
			{
				relative_y = 0;
			}
			if (relative_y >= height) 
			{
				relative_y = height - 1;
			}
			// Flatten our 2d coordinate
			int flatten_index = relative_x * width + relative_y;
			// Do convolution in each correspondence element in kernel with image
			r += (inPixels[flatten_index].x * filter[i]);
			g += (inPixels[flatten_index].y * filter[i]);
			b += (inPixels[flatten_index].z * filter[i]);
		}
		int flatten = x * width + y;
    outPixels[flatten] = make_uchar3(fabs(r), fabs(g), fabs(b));
	}
}
void Convolution(uchar3 * inPixels, int width, int height, float * filter, int filterWidth, 
        uchar3 * outPixels, dim3 blockSize=dim3(1, 1))
{
    size_t pixelsSize = width * height * sizeof(uchar3); 
    // Initilal in devices
    uchar3 * d_inPixels, * d_outPixels;
    float* device_filter;
    CHECK(hipMalloc(&d_inPixels, pixelsSize));
    CHECK(hipMemcpy(d_inPixels, inPixels, pixelsSize, hipMemcpyHostToDevice));

    CHECK(hipMalloc(&d_outPixels, pixelsSize));

    CHECK(hipMalloc(&device_filter, filterWidth * filterWidth * sizeof(float)));
    CHECK(hipMemcpy(device_filter, filter, filterWidth*filterWidth*sizeof(float), hipMemcpyHostToDevice));    
    
    
    dim3 gridSize((width-1)/blockSize.x + 1, (height-1)/blockSize.y + 1);
    int Cols = blockSize.x + filterWidth - 1;
    int Rows = blockSize.y + filterWidth - 1;
    // Call the kernel function
    Convolution_Kernel<<<gridSize, blockSize>>>(d_inPixels, width, height, device_filter, filterWidth, d_outPixels);
    hipDeviceSynchronize();
    CHECK(hipGetLastError());
    CHECK(hipMemcpy(outPixels, d_outPixels, pixelsSize, hipMemcpyDeviceToHost));
    
    CHECK(hipFree(d_inPixels));
    CHECK(hipFree(d_outPixels));
}
static double* getEnergy(uchar3* pixels, int width, int height, bool debug = 0)
{
    float * filter = (float *)malloc(3 * 3 * sizeof(float));
    //Convolution Sobel |Di/Dx|
    uchar3* Dx = (uchar3 *)malloc(width * height * sizeof(uchar3));
    filter[0]=1;filter[1]=0;filter[2]=-1;filter[3]=2;filter[4]=0;filter[5]=-2;filter[6]=1;filter[7]=0;filter[8]=-1;
    Convolution(pixels, width, height, filter, 3, Dx, dim3(32,32));
  
    //Convolution Sobel |Di/Dy|
    uchar3* Dy = (uchar3 *)malloc(width * height * sizeof(uchar3));
    filter[0]=1;filter[1]=2;filter[2]=1;filter[3]=0;filter[4]=0;filter[5]=0;filter[6]=-1;filter[7]=-2;filter[8]=-1;
    Convolution(pixels, width, height, filter, 3, Dy, dim3(32,32));
 
    if (debug == 1)
    {
      //Sum 3 channel and Dx and Dy to get energy
      uchar3* DebugEnergy = (uchar3*) malloc(width * height * sizeof(uchar3));
      
      for (int i = 0; i < width * height; i++)
      {
        int average = (((Dx[i].x) + (Dy[i].x))/2 + ((Dx[i].y) + (Dy[i].y))/2 + ((Dx[i].z) + (Dy[i].z))/2)/3;
        DebugEnergy[i].x = average;
        DebugEnergy[i].y = average;
        DebugEnergy[i].z = average;
      }
     
      writePnm(Dx, width, height, "Dx.pnm");
      writePnm(Dy, width, height, "Dy.pnm");
      writePnm(DebugEnergy, width, height, "energies.pnm");
      free(DebugEnergy);

    } 
    double* energies = (double*) malloc(width * height * sizeof(double));
    for (int i = 0; i < width * height; i++)
    {
      energies[i] = ((Dx[i].x)+(Dx[i].y)+(Dx[i].z))+((Dy[i].x)+(Dy[i].y)+(Dy[i].z));
    }

    free(Dx);
    free(Dy);
    free(filter);
    return energies;
  
}
int FindIndexOfMin(double* a, int left, int right)
{
    int index_of_min = left;
    double min_val = a[left];
    for(int i = left; i <= right; i++)
    {
        if (min_val > a[i])
           {
               min_val = a[i];
               index_of_min = i;
           }
    }
    return index_of_min;
}
__global__ void Calculate_Cumulative(double* energies,  int width, int height, double* cumulative, int* direction)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x; // This is index of column 0-->1024
  if (idx < width)
  {
      cumulative[idx] = energies[idx];
  }
  __syncthreads();
  for(int j = 1; j < height; j++)
  {
      if(idx == 0)
      {
          double mid = cumulative[(j-1)*width+idx];
          double right = cumulative[(j-1)*width+idx+1];
          if (mid < right)
          {
              cumulative[j*width+idx] = mid + energies[j*width+idx];
              direction[j*width+idx] = 1;
          }
          else
          {
              cumulative[j*width+idx] = right + energies[j*width+idx];
              direction[j*width+idx] = 2;
          }
      }
      else
      {
          if(idx == width -1)
          {
              double left = cumulative[(j-1)*width+idx-1];
              double mid = cumulative[(j-1)*width+idx];
              if (left < mid)
              {
                  cumulative[j*width+idx] = left + energies[j*width+idx];
                  direction[j*width+idx] = 0;
              }
              else
              {
                  cumulative[j*width+idx] = mid + energies[j*width+idx];
                  direction[j*width+idx] = 1;
              }
          }
          else
        {
             double left = cumulative[(j-1)*width+idx-1];
             double mid = cumulative[(j-1)*width+idx];
             double right = cumulative[(j-1)*width+idx+1];
             if (left <= mid && left <= right)
              {
                  cumulative[j*width+idx] = left + energies[j*width+idx];
                  direction[j*width+idx] = 0;
              }
             else
              { 
                  if(mid <= left && mid <= right)
                    {
                      cumulative[j*width+idx] = mid + energies[j*width+idx];
                      direction[j*width+idx] = 1;
                    }
                  else 
                   {
                       cumulative[j*width+idx] = right + energies[j*width+idx];
                       direction[j*width+idx] = 2;
                   }
              }
        }
      }
   __syncthreads();
  }
  
} 

static int*  get_route_of_minimum_cumulative_enery(const double* energies, int width, int height) {
    // cumulative enery
    double* cumulative =(double* ) malloc(width * height * sizeof(double));
    if (cumulative == NULL) return NULL;
    // 0: above-left   1: above    2: above-right
    int* direction =(int* ) calloc(width * height, sizeof(int));
    if (direction == NULL) return NULL;

    double * device_energies;
    double * device_cumulative;
    int* device_direction;

    CHECK(hipMalloc(&device_energies, width*height*sizeof(double)));
    CHECK(hipMemcpy(device_energies, energies, width*height*sizeof(double), hipMemcpyHostToDevice));

    CHECK(hipMalloc(&device_cumulative, width*height*sizeof(double)));
    CHECK(hipMalloc(&device_direction, width*height*sizeof(int)));



  
    dim3 blockSize=dim3(min(1024, width));
    dim3 gridSize = dim3(1);
    
 
  
    Calculate_Cumulative<<<gridSize, blockSize>>>(device_energies, width, height, device_cumulative, device_direction);
    hipDeviceSynchronize();
    CHECK(hipMemcpy(cumulative, device_cumulative, width*height*sizeof(double), hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(direction, device_direction, width*height*sizeof(int), hipMemcpyDeviceToHost));
    CHECK(hipGetLastError());
    CHECK(hipPeekAtLastError());

    
    int col = FindIndexOfMin(cumulative, (height - 1) * width + 0, (height - 1) * width + width-1) - ((height - 1) * width);
    free(cumulative);

    int* route = (int* )malloc(height * sizeof(int));
    if (route == NULL) return NULL;
    for (int r = height-1; r >= 0; r--) 
    {
        route[r] = col;
        int move = direction[r * width + col];
        if(direction[r * width + col] == 0) col--;
        else if(direction[r * width + col] == 2) col++;
    }
    free(direction);
    return route;
}
static void resize_image(uchar3* pixels, int width, int height, int num_of_column_deleted)
{
  uchar3* image_before = pixels;
  double* energies = (double*) malloc(height * width * sizeof(double));
  energies = getEnergy(pixels, width, height, 1);

 for (int i = 1; i <= num_of_column_deleted; i++)
 {
    int new_width = width - 1;
    uchar3* resize_remove_one_column_image = (uchar3 *) malloc(height*new_width*sizeof(uchar3));
    double* new_energies = (double* ) malloc(height * new_width * sizeof(double));
    int* min_route = get_route_of_minimum_cumulative_enery(energies, width, height);
    // Remove pixels
    for (int i = 0; i < height; i++) 
    {
      int index_of_resize_image = 0; // New j index after the offset
      for (int j = 0; j < width; j++) 
      {
          if (j != min_route[i])
          {
            resize_remove_one_column_image[i * new_width + index_of_resize_image] = image_before[i * width + j];
            index_of_resize_image++;
          }
      }
    }
    width--;
    new_energies = getEnergy(resize_remove_one_column_image, new_width, height);
    free(energies); free(min_route);
    image_before = resize_remove_one_column_image;
    energies = new_energies;
 }
 writePnm(image_before, width, height, "result.pnm");

}

int main(int argc, char ** argv)
{
  
  printDeviceInfo();
  printf("\n ------------------------------------------------------ \n");
  uchar3* inPixels;
  int width, height;
  readPnm(argv[1], width, height, inPixels);
  
  GpuTimer timer;
  timer.Start();
  resize_image(inPixels, width, height, atoi(argv[2]));
  timer.Stop();
  float total_time = timer.Elapsed();
  printf("Resize Image take: %f", total_time);
}